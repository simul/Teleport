#include "hip/hip_runtime.h"
// libavstream
// (c) Copyright 2018-2024 Simul.co

#include <cstdint>
#include <hip/hip_fp16.h>

surface<void, cudaSurfaceType2D> inputSurfaceRef;

__device__ float readDepth(int x, int y)
{
	half value;
	surf2Dread(&value, inputSurfaceRef, x << 1, y);
	return __half2float(value);
}

__device__ uint16_t readDepthU16(int x, int y)
{
	half value;
	surf2Dread(&value, inputSurfaceRef, x << 1, y);
	return __half2ushort_rd(value);
}

// Encode 16-bit depth value as YUV triplet in a way that minimizes error due to quantization and sub-sampling.
// See: "Adapting Standard Video Codecs for Depth Streaming", Fabrizo Pece, Jan Kautz, Tim Weyrich.
__device__ float3 encodeDepth(float Ld)
{
	const int np = 512;
	const int w  = 65536;
	const float p = float(np) / float(w);

	const float pDiv2 = p / 2.0f;
	const float pDiv4 = p / 4.0f;

	float Ha = fmod(Ld / pDiv2, 2.0f);
	if(Ha > 1.0f) {
		Ha = 2.0f - Ha;
	}

	float Hb = fmod((Ld - pDiv4) / pDiv2, 2.0f);
	if(Hb > 1.0f) {
		Hb = 2.0f - Hb;
	}

	return make_float3(Ld, Ha, Hb);
}

extern "C" __global__ void CopyPixels(uint32_t* pixels, int width, int height, int pitch)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= width || y >= height) {
		return;
	}

	surf2Dread(&pixels[y * (pitch >> 2) + x], inputSurfaceRef, x * 4, y);
}

extern "C" __global__ void CopyPixelsSwapRB(uchar4* pixels, int width, int height, int pitch)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= width || y >= height) {
		return;
	}

	uchar4 pixel;
	surf2Dread(&pixel, inputSurfaceRef, x * 4, y);
	pixels[y * (pitch >> 2) + x] = make_uchar4(pixel.z, pixel.y, pixel.x, pixel.w);
}

extern "C" __global__ void CopyPixels16(uint32_t* pixels, int width, int height, int pitch)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x >= width || y >= height) {

		return;
	}

	ushort4 p16;
	surf2Dread(&p16, inputSurfaceRef, x * 8, y);

	/* Aidan: Reduce to scale of 0-1023 (10-bit) from 0-65535 (16-bit) */
	p16.x /= 64;
	p16.y /= 64;
	p16.z /= 64;
	p16.w /= 64;

	uint32_t* pixel = &pixels[y * (pitch >> 2) + x];
	*pixel = uint32_t((p16.w << 30) | (p16.z << 20) | (p16.y << 10) | p16.x);
}

extern "C" __global__ void CopyPixels16SwapRB(uint32_t* pixels, int width, int height, int pitch)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x >= width || y >= height) {

		return;
	}

	ushort4 p16;
	surf2Dread(&p16, inputSurfaceRef, x * 8, y);

	/* Aidan: Reduce to scale of 0-1023 (10-bit) from 0-65535 (16-bit) */
	p16.x /= 64;
	p16.y /= 64;
	p16.z /= 64;
	p16.w /= 64; 

	uint32_t* pixel = &pixels[y * (pitch >> 2) + x];
	*pixel = uint32_t((p16.w << 30) | (p16.x << 20) | (p16.y << 10) | p16.z);
}

extern "C" __global__ void RGBAtoNV12(uint8_t* pixels, int width, int height, int pitch)
{
	// TODO: Implement
}

extern "C" __global__ void BGRAtoNV12(uint8_t* pixels, int width, int height, int pitch)
{
	// TODO: Implement
}

extern "C" __global__ void R16toNV12(uint8_t* pixels, int width, int height, int pitch, float remapNear, float remapFar)
{
	unsigned int x = 2 * (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int y = 2 * (blockDim.y * blockIdx.y + threadIdx.y);

	if(x >= width || y >= height) {
		return;
	}

	float inputD0 = readDepth(x  , y  );
	float inputD1 = readDepth(x+1, y  );
	float inputD2 = readDepth(x  , y+1);
	float inputD3 = readDepth(x+1, y+1);

	float depthRemapRange = remapFar - remapNear;
	if(depthRemapRange > 0.0f) {
		inputD0 = saturate((inputD0 - remapNear) / depthRemapRange);
		inputD1 = saturate((inputD1 - remapNear) / depthRemapRange);
		inputD2 = saturate((inputD2 - remapNear) / depthRemapRange);
		inputD3 = saturate((inputD3 - remapNear) / depthRemapRange);
	}

	float3 D0 = encodeDepth(inputD0);
	float3 D1 = encodeDepth(inputD1);
	float3 D2 = encodeDepth(inputD2);
	float3 D3 = encodeDepth(inputD3);

	float U = 0.25f * (D0.y + D1.y + D2.y + D3.y);
	float V = 0.25f * (D0.z + D1.z + D2.z + D3.z);

	pixels[(y  ) * pitch + x  ] = D0.x * 255;
	pixels[(y  ) * pitch + x+1] = D1.x * 255;
	pixels[(y+1) * pitch + x  ] = D2.x * 255;
	pixels[(y+1) * pitch + x+1] = D3.x * 255;

	const int chromaOffset = pitch * height;
	pixels[chromaOffset + (y >> 1) * pitch + x  ] = U * 255;
	pixels[chromaOffset + (y >> 1) * pitch + x+1] = V * 255;
}

