#include "hip/hip_runtime.h"
// libavstream
// (c) Copyright 2018-2024 Simul.co

#include <cstdint>
#include <hip/hip_fp16.h>

surface<void, cudaSurfaceType2D> outputSurfaceRef;

__device__ const float YUVtoRGB_ITU709[3][3] = {
	{ 1.0f,  0.0f,       1.5748f   },
	{ 1.0f, -0.187324f, -0.468124f },
	{ 1.0f,  1.8556f  ,  0.0f      },
};

__device__ const float YUVtoRGB_BT709[3][3] = {
	{ 1.16438f,  0.0f,  1.83367f   },
	{ 1.16438f, -0.218117f, -0.545076f },
	{ 1.16438f,  2.16063f,  0.0f      },
};

__device__ const float YUVtoRGB_BT2020[3][3] = {
	{ 1.16893f,  0.0f,       1.72371f   },
	{ 1.16893f, -0.192351f, -0.667873f },
	{ 1.16893f,  2.19923f  ,  0.0f      },
};

template<class T>
__device__ static T Clamp(T x, T lower, T upper) {
	return x < lower ? lower : (x > upper ? upper : x);
}

template<class YuvUnit>
__device__ inline uchar3 YuvToRgbForPixel(YuvUnit y, YuvUnit u, YuvUnit v, int matrixType) 
{
	const int
		low = 1 << (sizeof(YuvUnit) * 8 - 4),
		mid = 1 << (sizeof(YuvUnit) * 8 - 1);
	float fy = (int)y - low, fu = (int)u - mid, fv = (int)v - mid;
	const float maxf = (1 << sizeof(YuvUnit) * 8) - 1.0f;
	YuvUnit r, g, b = 0;
	if (matrixType == 0)
	{
		r = (YuvUnit)Clamp(YUVtoRGB_BT709[0][0] * fy + YUVtoRGB_BT709[0][1] * fu + YUVtoRGB_BT709[0][2] * fv, 0.0f, maxf);
		g = (YuvUnit)Clamp(YUVtoRGB_BT709[1][0] * fy + YUVtoRGB_BT709[1][1] * fu + YUVtoRGB_BT709[1][2] * fv, 0.0f, maxf);
		b = (YuvUnit)Clamp(YUVtoRGB_BT709[2][0] * fy + YUVtoRGB_BT709[2][1] * fu + YUVtoRGB_BT709[2][2] * fv, 0.0f, maxf);
	}
	else if (matrixType == 4)
	{
		r = (YuvUnit)Clamp(YUVtoRGB_BT2020[0][0] * fy + YUVtoRGB_BT2020[0][1] * fu + YUVtoRGB_BT2020[0][2] * fv, 0.0f, maxf);
		g = (YuvUnit)Clamp(YUVtoRGB_BT2020[1][0] * fy + YUVtoRGB_BT2020[1][1] * fu + YUVtoRGB_BT2020[1][2] * fv, 0.0f, maxf);
		b = (YuvUnit)Clamp(YUVtoRGB_BT2020[2][0] * fy + YUVtoRGB_BT2020[2][1] * fu + YUVtoRGB_BT2020[2][2] * fv, 0.0f, maxf);
	}

	uchar3 rgb;
	const int nShift = abs((int)sizeof(YuvUnit) - (int)sizeof(rgb.x)) * 8;
	if (sizeof(YuvUnit) >= sizeof(rgb.x)) 
	{
		rgb.x = r >> nShift;
		rgb.y = g >> nShift;
		rgb.z = b >> nShift;
	}
	else 
	{
		rgb.x = r << nShift;
		rgb.y = g << nShift;
		rgb.z = b << nShift;
	}
	
	return make_uchar3(rgb.x, rgb.y, rgb.z);
}

template<class YuvUnitx2>
__device__ void inline YuvToRgb(const uint8_t* frame, int width, int height, int pitch, int matrixType, bool isABGR)
{
	unsigned int x = 2 * (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int y = 2 * (blockDim.y * blockIdx.y + threadIdx.y);

	if (x + 1 >= width || y + 1>= height) {
		return;
	}

	uint8_t* pSrc = (uint8_t *)&frame[x * sizeof(YuvUnitx2) / 2 + y * pitch];
	// 4 Ys
	YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
	YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + pitch);
	// U and V components
	YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (height - y / 2) * pitch);

	uchar4 p0, p1, p2, p3;

	if (isABGR)
	{
		p0 = packABGR8(YuvToRgbForPixel(l0.x, ch.x, ch.y, matrixType));
		p1 = packABGR8(YuvToRgbForPixel(l0.y, ch.x, ch.y, matrixType));
		p2 = packABGR8(YuvToRgbForPixel(l1.x, ch.x, ch.y, matrixType));
		p3 = packABGR8(YuvToRgbForPixel(l1.y, ch.x, ch.y, matrixType));
	}
	else
	{
		p0 = packRGBA8(YuvToRgbForPixel(l0.x, ch.x, ch.y, matrixType));
		p1 = packRGBA8(YuvToRgbForPixel(l0.y, ch.x, ch.y, matrixType));
		p2 = packRGBA8(YuvToRgbForPixel(l1.x, ch.x, ch.y, matrixType));
		p3 = packRGBA8(YuvToRgbForPixel(l1.y, ch.x, ch.y, matrixType));
	}

	surf2Dwrite(p0, outputSurfaceRef, (x) * 4, (y), hipBoundaryModeZero);
	surf2Dwrite(p1, outputSurfaceRef, (x + 1) * 4, (y), hipBoundaryModeZero);
	surf2Dwrite(p2, outputSurfaceRef, (x) * 4, (y + 1), hipBoundaryModeZero);
	surf2Dwrite(p3, outputSurfaceRef, (x + 1) * 4, (y + 1), hipBoundaryModeZero);
}

template<class YuvUnitx2>
__device__ void inline Yuv444ToRgb(const uint8_t* frame, int width, int height, int pitch, int matrixType, bool isABGR)
{
	unsigned int x = 2 * (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int y = (blockDim.y * blockIdx.y + threadIdx.y);

	if (x + 1 >= width || y >= height) {
		return;
	}

	uint8_t* pSrc = (uint8_t *)&frame[x * sizeof(YuvUnitx2) / 2 + y * pitch];
	// 4 Ys
	YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
	// U and V components
	YuvUnitx2 ch1 = *(YuvUnitx2 *)(pSrc + (height * pitch));
	YuvUnitx2 ch2 = *(YuvUnitx2 *)(pSrc + (2 * height * pitch));

	uchar4 p0, p1;

	if (isABGR)
	{
		p0 = packABGR8(YuvToRgbForPixel(l0.x, ch1.x, ch2.x, matrixType));
		p1 = packABGR8(YuvToRgbForPixel(l0.y, ch1.y, ch2.y, matrixType));
	}
	else
	{
		p0 = packRGBA8(YuvToRgbForPixel(l0.x, ch1.x, ch2.x, matrixType));
		p1 = packRGBA8(YuvToRgbForPixel(l0.y, ch1.y, ch2.y, matrixType));
	}

	surf2Dwrite(p0, outputSurfaceRef, (x) * 4, (y), hipBoundaryModeZero);
	surf2Dwrite(p1, outputSurfaceRef, (x + 1) * 4, (y), hipBoundaryModeZero);
}

template<class YuvUnit>
__device__ inline unsigned char YuvToAlphaForPixel(YuvUnit y)
{
	const int low = 1 << (sizeof(YuvUnit) * 8 - 4);
	float fy = (int)y - low;
	const float maxf = (1 << sizeof(YuvUnit) * 8) - 1.0f;

	// uvs not needed. alpha is stored in the y channel only.
	YuvUnit a = (YuvUnit)Clamp(YUVtoRGB_BT709[0][0] * fy, 0.0f, maxf);
	return a;
}

template<class YuvUnitx2>
__device__ void inline YuvToAlpha(const uint8_t* frame, int width, int height, int pitch, bool isABGR)
{
	unsigned int x = 2 * (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int y = 2 * (blockDim.y * blockIdx.y + threadIdx.y);

	if (x + 1 >= width || y + 1 >= height) {
		return;
	}

	uint8_t* pSrc = (uint8_t*)&frame[x * sizeof(YuvUnitx2) / 2 + y * pitch];
	// 4 Ys
	YuvUnitx2 l0 = *(YuvUnitx2*)pSrc;
	YuvUnitx2 l1 = *(YuvUnitx2*)(pSrc + pitch);

	uchar4 p0, p1, p2, p3;

	surf2Dread(&p0, outputSurfaceRef, x * 4, y);
	surf2Dread(&p1, outputSurfaceRef, (x + 1) * 4, y);
	surf2Dread(&p2, outputSurfaceRef, x * 4, y + 1);
	surf2Dread(&p3, outputSurfaceRef, (x + 1) * 4, y + 1);

	if (isABGR)
	{
		p0.x = YuvToAlphaForPixel(l0.x);
		p1.x = YuvToAlphaForPixel(l0.y);
		p2.x = YuvToAlphaForPixel(l1.x);
		p3.x = YuvToAlphaForPixel(l1.y);
	}
	else
	{
		p0.w = YuvToAlphaForPixel(l0.x);
		p1.w = YuvToAlphaForPixel(l0.y);
		p2.w = YuvToAlphaForPixel(l1.x);
		p3.w = YuvToAlphaForPixel(l1.y);
	}

	surf2Dwrite(p0, outputSurfaceRef, (x) * 4, (y), hipBoundaryModeZero);
	surf2Dwrite(p1, outputSurfaceRef, (x + 1) * 4, (y), hipBoundaryModeZero);
	surf2Dwrite(p2, outputSurfaceRef, (x) * 4, (y + 1), hipBoundaryModeZero);
	surf2Dwrite(p3, outputSurfaceRef, (x + 1) * 4, (y + 1), hipBoundaryModeZero);
}

__device__ uchar4 packRGBA8(uchar3 value)
{
	return make_uchar4(value.x, value.y, value.z, 255);
}

__device__ uchar4 packABGR8(uchar3 value)
{
	return make_uchar4(255, value.z, value.y, value.x);
}

// Decode 16-bit depth value from quantized & sub-sampled YUV triplet.
// See: "Adapting Standard Video Codecs for Depth Streaming", Fabrizo Pece, Jan Kautz, Tim Weyrich.
__device__ float decodeDepth(float Ld, float Ha, float Hb)
{
	const int np = 512;
	const int w  = 65536;
	const float p = float(np) / float(w);

	const float pDiv2 = p / 2.0f;
	const float pDiv4 = p / 4.0f;
	const float pDiv8 = p / 8.0f;
	
	int   mL = __float2int_rd(4.0f * (Ld / p) - 0.5f) % 4;
	float L0 = Ld - fmod(Ld - pDiv8, p) + pDiv4 * mL - pDiv8;

	float deltaH;
	switch(mL) {
	case 0: deltaH = pDiv2 * Ha; break;
	case 1: deltaH = pDiv2 * Hb; break;
	case 2: deltaH = pDiv2 * (1.0f - Ha); break;
	case 3: deltaH = pDiv2 * (1.0f - Hb); break;
	}

	return L0 + deltaH;
}

extern "C" __global__ void NV12toRGBA(const uint8_t* frame, int width, int height, int pitch)
{
	YuvToRgb<uchar2>(frame, width, height, pitch, 0, false);
}

extern "C" __global__ void NV12toABGR(const uint8_t* frame, int width, int height, int pitch)
{
	YuvToRgb<uchar2>(frame, width, height, pitch, 0, true);
}

extern "C" __global__ void AlphaNV12toRGBA(const uint8_t * frame, int width, int height, int pitch)
{
	YuvToAlpha<uchar2>(frame, width, height, pitch, false);
}

extern "C" __global__ void AlphaNV12toABGR(const uint8_t * frame, int width, int height, int pitch)
{
	YuvToAlpha<uchar2>(frame, width, height, pitch, true);
}

extern "C" __global__ void P016toRGBA(const uint8_t* frame, int width, int height, int pitch)
{
	YuvToRgb<ushort2>(frame, width, height, pitch, 4, false);
}

extern "C" __global__ void P016toABGR(const uint8_t* frame, int width, int height, int pitch)
{
	YuvToRgb<ushort2>(frame, width, height, pitch, 4, true);
}

extern "C" __global__ void YUV444toRGBA(const uint8_t* frame, int width, int height, int pitch)
{
	Yuv444ToRgb<uchar2>(frame, width, height, pitch, 0, false);
}

extern "C" __global__ void YUV444toABGR(const uint8_t* frame, int width, int height, int pitch)
{
	Yuv444ToRgb<uchar2>(frame, width, height, pitch, 0, true);
}

extern "C" __global__ void YUV444P16toRGBA(const uint8_t* frame, int width, int height, int pitch)
{
	Yuv444ToRgb<ushort2>(frame, width, height, pitch, 4, false);
}

extern "C" __global__ void YUV444P16toABGR(const uint8_t* frame, int width, int height, int pitch)
{
	Yuv444ToRgb<ushort2>(frame, width, height, pitch, 4, true);
}

extern "C" __global__ void NV12toR16(const uint8_t* frame, int width, int height, int pitch)
{
	unsigned int x = 2 * (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int y = 2 * (blockDim.y * blockIdx.y + threadIdx.y);

	if (x >= width || y >= height) {
		return;
	}

	float Y0 = frame[(y)* pitch + x] / 255.0f;
	float Y1 = frame[(y)* pitch + x + 1] / 255.0f;
	float Y2 = frame[(y + 1) * pitch + x] / 255.0f;
	float Y3 = frame[(y + 1) * pitch + x + 1] / 255.0f;

	const int chromaOffset = pitch * height;
	float U = frame[chromaOffset + (y >> 1) * pitch + x] / 255.0f;
	float V = frame[chromaOffset + (y >> 1) * pitch + x + 1] / 255.0f;

	surf2Dwrite(__float2half(decodeDepth(Y0, U, V)), outputSurfaceRef, (x) << 1, (y), hipBoundaryModeZero);
	surf2Dwrite(__float2half(decodeDepth(Y1, U, V)), outputSurfaceRef, (x + 1) << 1, (y), hipBoundaryModeZero);
	surf2Dwrite(__float2half(decodeDepth(Y2, U, V)), outputSurfaceRef, (x) << 1, (y + 1), hipBoundaryModeZero);
	surf2Dwrite(__float2half(decodeDepth(Y3, U, V)), outputSurfaceRef, (x + 1) << 1, (y + 1), hipBoundaryModeZero);
}
